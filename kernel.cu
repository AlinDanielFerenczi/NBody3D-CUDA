#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"
#define DllExport __declspec(dllexport)
#include <stdio.h>
#include <string>

class Vector3D
{
public:
    float X;
    float Y;
    float Z;
};

extern "C"
{
    DllExport hipError_t calculatePosition(Vector3D* positions, Vector3D* velocities, Vector3D* accelerations, float* masses, float timeStep, int size);
    hipError_t calculatePosition(Vector3D* positions, Vector3D* velocities, Vector3D* accelerations, float* masses, float timeStep, int size);
}

__global__ void calculateVelocityKernel(Vector3D* velocities, Vector3D* accelerations, float timeStep)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    float halfStep = timeStep / 2;
    float multipliedX = accelerations[i].X * halfStep + velocities[i].X;
    float multipliedY = accelerations[i].Y * halfStep + velocities[i].Y;
    float multipliedZ = accelerations[i].Z * halfStep + velocities[i].Z;

    Vector3D result;
    result.X = multipliedX;
    result.Y = multipliedY;
    result.Z = multipliedZ;

    velocities[i] = result;
}

//__device__ unsigned long long totThr = 0;

__global__ void calculatePositionKernel(Vector3D *positions, Vector3D *velocities, float timeStep)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    float multipliedX = velocities[i].X * timeStep + positions[i].X;
    float multipliedY = velocities[i].Y * timeStep + positions[i].Y;
    float multipliedZ = velocities[i].Z * timeStep + positions[i].Z;

    Vector3D result;
    result.X = multipliedX;
    result.Y = multipliedY;
    result.Z = multipliedZ;

    positions[i] = result;
}

__global__ void calculateAccelerationKernel(Vector3D* positions, float* masses, Vector3D* accelerations) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    float diffX = positions[j].X - positions[i].X;
    float diffY = positions[j].Y - positions[i].Y;
    float diffZ = positions[j].Z - positions[i].Z;

    float dist = sqrtf(diffX * diffX + diffY * diffY + diffZ * diffZ);
    float F = (1 * masses[i] * masses[j]) / (dist * dist + 0.1 * 0.1);

    //atomicAdd(&totThr, 1);

    atomicAdd(&accelerations[i].X, diffX * F * dist);
    atomicAdd(&accelerations[i].Y, diffY * F * dist);
    atomicAdd(&accelerations[i].Z, diffZ * F * dist);
}

__global__ void initAccelerationsKernel(Vector3D* accelerations)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    Vector3D newVector;
    newVector.X = 0;
    newVector.Y = 0;
    newVector.Z = 0;
    accelerations[i] = newVector;
}

hipError_t calculatePosition(Vector3D* positions, Vector3D* velocities, Vector3D* accelerations, float* masses, float timeStep, int size)
{
    FILE* fptr;
    fptr = fopen("fileopen.txt", "w");
    Vector3D* dev_positions = 0;
    Vector3D* dev_velocities = 0;
    Vector3D* dev_accelerations = 0;
    float* dev_masses = 0;
    hipError_t cudaStatus;

    /*for (int i = 0; i < size; i++)
    {
        fprintf(fptr,
            "X for object %f with position: %f velocity: %f mass: %f\n",
            i, positions[i].X, velocities[i].X, masses[i]
        );
        fprintf(fptr,
            "Y for object %f with position: %f velocity: %f mass: %f\n",
            i, positions[i].Y, velocities[i].Z, masses[i]
        );
        fprintf(fptr,
            "Z for object %f with position: %f velocity: %f mass: %f\n\n",
            i, positions[i].Z, velocities[i].Y, masses[i]
        );
    }*/

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .

    cudaStatus = hipMalloc((void**)&dev_positions, size * sizeof(Vector3D));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_velocities, size * sizeof(Vector3D));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_masses, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_accelerations, size * sizeof(Vector3D));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_positions, positions, size * sizeof(Vector3D), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_velocities, velocities, size * sizeof(Vector3D), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_accelerations, accelerations, size * sizeof(Vector3D), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_masses, masses, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    dim3 block(32, 32);
    dim3 grid;
    grid.x = (size + block.x - 1) / block.x;
    grid.y = (size + block.y - 1) / block.y;
    dim3 simpleGrid;
    simpleGrid = (size + block.x - 1) / block.x;
    calculateVelocityKernel <<<simpleGrid, block>>> (dev_velocities, dev_accelerations, timeStep);
    calculatePositionKernel <<<simpleGrid, block>>> (dev_positions, dev_velocities, timeStep);
    initAccelerationsKernel <<<simpleGrid, block>>> (dev_accelerations);
    calculateAccelerationKernel <<<grid, block>>> (dev_positions, dev_masses, dev_accelerations);
    calculateVelocityKernel <<<simpleGrid, block>>> (dev_velocities, dev_accelerations, timeStep);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(positions, dev_positions, size * sizeof(Vector3D), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(velocities, dev_velocities, size * sizeof(Vector3D), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //unsigned long long total;
    //hipMemcpyFromSymbol(&total, HIP_SYMBOL(totThr), sizeof(unsigned long long));
    //fprintf(fptr, "Total threads counted: %lu\n", total);

Error:
    hipFree(dev_positions);
    hipFree(dev_velocities);
    hipFree(dev_accelerations);
    hipFree(dev_masses);
    //fclose(fptr);

    return cudaStatus;
}